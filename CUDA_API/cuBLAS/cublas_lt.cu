#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h> // this is secifically when we use "half" data type
#include <hipblaslt.h>
#include <time.h>

// to run this code: nvcc cublas_lt.cu -o cublas_lt -L/usr/local/cuda/lib64 -lcublas -lcublasLt -lcudart && ./cublas_lt

using namespace std;

#define CUDA_CHECK(err) gpuAssert(err, __FILE__, __LINE__)
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort=true){
  if(code != hipSuccess){
    fprintf(stderr, "GPUAssert: %s %s %d\n", hipGetErrorString(code), file, line);
    if(abort) exit(code);
  }
}

#define CUBLAS_CHECK(err) cublasAssert(err, __FILE__, __LINE__)
inline void cublasAssert(hipblasStatus_t code, const char* file, int line, bool abort=true){
  if(code != HIPBLAS_STATUS_SUCCESS){
    fprintf(stderr, "CUBLASAssert: %d %s %d\n", code, file, line);
    if(abort) exit(code);
  }
}

double get_time(){
  struct timespec ts;
  clock_gettime(CLOCK_MONOTONIC, &ts);
  return ts.tv_sec + ts.tv_nsec * 1e-9;
}

int main(){
  int M=4, K=4, N=4;

  float h_a[M*K] = {
      1.0f, 2.0f, 3.0f, 4.0f,
      5.0f, 6.0f, 7.0f, 8.0f,
      9.0f, 10.0f, 11.0f, 12.0f,
      13.0f, 14.0f, 15.0f, 16.0f
  };

  float h_b[K*N] = {
      1.0f, 2.0f, 3.0f, 4.0f,
      5.0f, 6.0f, 7.0f, 8.0f,
      9.0f, 10.0f, 11.0f, 12.0f,
      17.0f, 18.0f, 19.0f, 20.0f
  };

  // float h_c_cpu[  M*N] = {0};
  float h_c_gpu_fp32[M*N] = {0};
  float  h_c_gpu_fp16[M*N] = {0};

  double start, end;

  // first fp32
  float *d_a_fp32, *d_b_fp32, *d_c_fp32;
  start = get_time();
  CUDA_CHECK(hipMalloc((void**)&d_a_fp32, M*K*sizeof(float)));
  end = get_time();
  printf("Time for hipMalloc d_a_fp32: %f\n", end - start);
  start = get_time();
  CUDA_CHECK(hipMalloc((void**)&d_b_fp32, K*N*sizeof(float)));
  end = get_time();
  printf("Time for hipMalloc d_b_fp32: %f\n", end - start);
  start = get_time();
  CUDA_CHECK(hipMalloc((void**)&d_c_fp32, M*N*sizeof(float)));
  end = get_time();
  printf("Time for hipMalloc d_c_fp32: %f\n", end - start);

  start = get_time();
  CUDA_CHECK(hipMemcpy(d_a_fp32, h_a, M*K*sizeof(float), hipMemcpyHostToDevice));
  end = get_time();
  printf("Time for hipMemcpy d_a_fp32: %f\n", end - start);
  start = get_time();
  CUDA_CHECK(hipMemcpy(d_b_fp32, h_b, K*N*sizeof(float), hipMemcpyHostToDevice));
  end = get_time();
  printf("Time for hipMemcpy d_b_fp32: %f\n", end - start);

  start = get_time();
  hipblasLtHandle_t handle;
  CUBLAS_CHECK(hipblasLtCreate(&handle));
  end = get_time();
  printf("Time for hipblasLtCreate: %f\n", end - start);

  // setting up matrix descriptors now
  start = get_time();
  hipblasLtMatrixLayout_t mata_fp32, matb_fp32, matc_fp32;
  CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&mata_fp32, HIP_R_32F, K, M, K));
  CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&matb_fp32, HIP_R_32F, N, K, N));
  CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&matc_fp32, HIP_R_32F, N, M, N));
  end = get_time();
  printf("Time for hipblasLtMatrixLayoutCreate (all 3): %f\n", end - start);

  // now mat mul descriptors
  hipblasLtMatmulDesc_t matmul_desc_fp32;
  CUBLAS_CHECK(hipblasLtMatmulDescCreate(&matmul_desc_fp32, HIPBLAS_COMPUTE_32F, HIP_R_32F)); // name, compute type, scale type

  hipblasOperation_t transa = HIPBLAS_OP_N;
  hipblasOperation_t transb = HIPBLAS_OP_N;

  CUBLAS_CHECK(hipblasLtMatmulDescSetAttribute(matmul_desc_fp32, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
  CUBLAS_CHECK(hipblasLtMatmulDescSetAttribute(matmul_desc_fp32, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb)));

  float alpha = 1.0f, beta = 0.0f;

  start = get_time();
  CUBLAS_CHECK(hipblasLtMatmul(handle, matmul_desc_fp32, &alpha, d_b_fp32, matb_fp32, d_a_fp32, mata_fp32, &beta, d_c_fp32, matc_fp32, d_c_fp32, matc_fp32, NULL, NULL, 0, 0));
  end = get_time();
  printf("Time for hipblasLtMatmul (fp32): %f\n", end - start);
  start = get_time();
  CUDA_CHECK(hipMemcpy(h_c_gpu_fp32, d_c_fp32, M*N*sizeof(float), hipMemcpyDeviceToHost));
  end = get_time();
  printf("Time for hipMemcpy result (fp32): %f\n", end - start);

  // now fp16
  half h_a_h[M*K], h_b_h[K*N];
  start = get_time();
  for(int i=0; i<M*K; i++) h_a_h[i] = __float2half(h_a[i]);
  for(int i=0; i<K*N; i++) h_b_h[i] = __float2half(h_b[i]);
  end = get_time();
  printf("Time for half conversion: %f\n", end - start);

  half *d_a_h, *d_b_h, *d_c_h;
  CUDA_CHECK(hipMalloc((void**)&d_a_h, M*K*sizeof(half)));
  CUDA_CHECK(hipMalloc((void**)&d_b_h, K*N*sizeof(half)));
  CUDA_CHECK(hipMalloc((void**)&d_c_h, M*N*sizeof(half)));
  start = get_time();
  CUDA_CHECK(hipMemcpy(d_a_h, h_a_h, M*K*sizeof(half), hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(d_b_h, h_b_h, K*N*sizeof(half), hipMemcpyHostToDevice));
  end = get_time();
  printf("Time for hipMemcpy (fp16): %f\n", end - start);

  hipblasLtMatrixLayout_t mata_h, matb_h, matc_h;
  CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&mata_h, HIP_R_16F, K, M, K));
  CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&matb_h, HIP_R_16F, N, K, N));
  CUBLAS_CHECK(hipblasLtMatrixLayoutCreate(&matc_h, HIP_R_16F, N, M, N));
  start = get_time();
  hipblasLtMatmulDesc_t matmul_desc_h;
  CUBLAS_CHECK(hipblasLtMatmulDescCreate(&matmul_desc_h, HIPBLAS_COMPUTE_16F, HIP_R_16F));
  end = get_time();
  printf("Time for hipblasLtMatmulDescCreate (fp16): %f\n", end - start); // name, compute type, scale type

  CUBLAS_CHECK(hipblasLtMatmulDescSetAttribute(matmul_desc_h, HIPBLASLT_MATMUL_DESC_TRANSA, &transa, sizeof(transa)));
  CUBLAS_CHECK(hipblasLtMatmulDescSetAttribute(matmul_desc_h, HIPBLASLT_MATMUL_DESC_TRANSB, &transb, sizeof(transb)));

  half alpha_h = __float2half(alpha), beta_h = __float2half(beta);
  start = get_time();
  CUBLAS_CHECK(hipblasLtMatmul(handle, matmul_desc_h, &alpha_h, d_b_h, matb_h, d_a_h, mata_h, &beta_h, d_c_h, matc_h, d_c_h, matc_h, NULL, NULL, 0, 0));
  end = get_time();
  printf("Time for hipblasLtMatmul (fp16): %f\n", end - start);
  start = get_time();
  CUDA_CHECK(hipMemcpy(h_c_gpu_fp16, d_c_h, M*N*sizeof(half), hipMemcpyDeviceToHost));
  end = get_time();
  printf("Time for hipMemcpy result (fp16): %f\n", end - start);

  // now printing results
  printf("CUBLASLt fp32 Result:\n");
  for(int i=0; i<M; i++){
    for(int j=0; j<N; j++){
      printf("%0.2f ", h_c_gpu_fp32[i*N + j]);
    }
    printf("\n");
  }

  printf("\nCUBLASLt fp16 Result:\n");
  for(int i=0; i<M; i++){
    for(int j=0; j<N; j++){
      printf("%0.2f ", __half2float(h_c_gpu_fp16[i*N + j]));
    }
    printf("\n");
  }

  // free all resources
  CUDA_CHECK(hipFree(d_a_fp32));
  CUDA_CHECK(hipFree(d_b_fp32));
  CUDA_CHECK(hipFree(d_c_fp32));
  CUDA_CHECK(hipFree(d_a_h));
  CUDA_CHECK(hipFree(d_b_h));
  CUDA_CHECK(hipFree(d_c_h));
  CUBLAS_CHECK(hipblasLtDestroy(handle));
  CUBLAS_CHECK(hipblasLtMatrixLayoutDestroy(mata_fp32));
  CUBLAS_CHECK(hipblasLtMatrixLayoutDestroy(matb_fp32));
  CUBLAS_CHECK(hipblasLtMatrixLayoutDestroy(matc_fp32));
  CUBLAS_CHECK(hipblasLtMatrixLayoutDestroy(mata_h));
  CUBLAS_CHECK(hipblasLtMatrixLayoutDestroy(matb_h));
  CUBLAS_CHECK(hipblasLtMatrixLayoutDestroy(matc_h));
  CUBLAS_CHECK(hipblasLtMatmulDescDestroy(matmul_desc_fp32));
  CUBLAS_CHECK(hipblasLtMatmulDescDestroy(matmul_desc_h));

  return 0;
}