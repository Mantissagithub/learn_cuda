#include <iostream>
#include <hip/hip_runtime.h>

#define NUM_STREAMS 4

__global__ void vectorAdd(float *a, float *b, float* c, int n){
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    if(idx < n){
        c[idx] = a[idx] + b[idx];
    }
}

int main(){
    int size = 1000000;
    int bytes = size * sizeof(float);

    hipStream_t streams[NUM_STREAMS];
    for(int i=0;i<NUM_STREAMS;i++){
        hipStreamCreate(&streams[i]);
    }

    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);

    hipEvent_t memcpyh2d_events[NUM_STREAMS];
    hipEvent_t kernel_events[NUM_STREAMS];
    hipEvent_t memcpyd2h_events[NUM_STREAMS];

    for(int i=0;i<NUM_STREAMS;i++){
        hipEventCreate(&memcpyh2d_events[i]);
        hipEventCreate(&kernel_events[i]);
        hipEventCreate(&memcpyd2h_events[i]);
    }

    float *h_a, *h_b, *h_c;
    hipHostMalloc((void**)&h_a, bytes, hipHostMallocDefault);
    hipHostMalloc((void**)&h_b, bytes, hipHostMallocDefault);
    hipHostMalloc((void**)&h_c, bytes, hipHostMallocDefault);

    for(int i = 0; i < size; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    float *d_a[NUM_STREAMS], *d_b[NUM_STREAMS], *d_c[NUM_STREAMS];\
    int streamSize = size / NUM_STREAMS;
    int streamBytes = streamSize * sizeof(float);

    for(int i=0;i<NUM_STREAMS;i++){
        hipMalloc(&d_a[i], bytes);
        hipMalloc(&d_b[i], bytes);
        hipMalloc(&d_c[i], bytes);
    }

    hipEventRecord(startEvent);

    for(int i=0;i<NUM_STREAMS;i++){
        int offset = i * streamSize;

        hipMemcpyAsync(d_a[i], h_a + offset, streamBytes, hipMemcpyHostToDevice, streams[i]);
        hipMemcpyAsync(d_b[i], h_b + offset, streamBytes, hipMemcpyHostToDevice, streams[i]);
        hipEventRecord(memcpyh2d_events[i], streams[i]);

        dim3 block(256);
        dim3 grid((streamSize + block.x - 1)/block.x);

        vectorAdd<<<grid, block, 0, streams[i]>>>(d_a[i], d_b[i], d_c[i], streamSize);
        hipEventRecord(kernel_events[i], streams[i]);

        hipMemcpyAsync(h_c + offset, d_c[i], streamBytes, hipMemcpyDeviceToHost, streams[i]);
        hipEventRecord(memcpyd2h_events[i], streams[i]);
    }


    hipEventRecord(stopEvent);

    hipEventSynchronize(stopEvent);

    float totalTime;
    hipEventElapsedTime(&totalTime, startEvent, stopEvent);

    printf("Total time with events and streams: %f seconds\n", totalTime / 1000.0f);

    for(int i = 0; i < NUM_STREAMS; i++) {
        float h2dTime, kernelTime, d2hTime;
        
        hipEventElapsedTime(&h2dTime, startEvent, memcpyh2d_events[i]);
         
        hipEventElapsedTime(&kernelTime, startEvent, kernel_events[i]);
        
        hipEventElapsedTime(&d2hTime, startEvent, memcpyd2h_events[i]);
        
        printf("Stream %d - H2D: %.3f ms, Kernel: %.3f ms, D2H: %.3f ms\n", i, h2dTime, kernelTime, d2hTime);
    }

    hipEventDestroy(startEvent);
    hipEventDestroy(stopEvent);

    for(int i=0;i<NUM_STREAMS;i++){
        hipEventDestroy(memcpyh2d_events[i]);
        hipEventDestroy(kernel_events[i]);
        hipEventDestroy(memcpyd2h_events[i]);
    }

    for(int i=0;i<NUM_STREAMS;i++){
        hipFree(d_a[i]); hipFree(d_b[i]); hipFree(d_c[i]);
        hipStreamDestroy(streams[i]);
    }

    hipHostFree(h_a); hipHostFree(h_b); hipHostFree(h_c);

    return 0;
}