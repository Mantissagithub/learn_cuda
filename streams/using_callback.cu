#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
#include <time.h>

using namespace std;

#define CUDA_CHECK(err) gpuAssert((err), __FILE__, __LINE__)
inline void gpuAssert(hipError_t code, const char* file, int line, bool abort=true){
  if(code != hipSuccess){
    fprintf(stderr, "GPUAssert: %s %s %d\n", hipGetErrorString(code), file, line);
    if(abort) exit(code);
  }
}

__global__ void kernel2(float* data, int n){
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if(idx < n){
    data[idx] *= 2.0f;
  }
}

__global__ void kernel1(float* data, int n){
  int idx = blockDim.x * blockIdx.x + threadIdx.x;
  if(idx < n){
    data[idx] += 1.0f;
  }
}

void CUDART_CB streamCallback(hipStream_t stream, hipError_t status, void *UserData){
  printf("Stream callback, operation completed\n");
}

int main(){
  int N = 10000;
  size_t size = N * sizeof(float);
  float *h_data, *d_data;
  hipStream_t stream1, stream2;
  hipEvent_t event;

  // cout<<event<<endl;

  CUDA_CHECK(hipHostMalloc(&h_data, size, hipHostMallocDefault)); // pinned mem
  CUDA_CHECK(hipMalloc(&d_data, size));

  for(int i=0;i<N;i++){
    h_data[i] = (float)i*2.0f;
  }

  int leastPriority, greaterPriority;
  CUDA_CHECK(hipDeviceGetStreamPriorityRange(&leastPriority, &greaterPriority));
  CUDA_CHECK(hipStreamCreateWithPriority(&stream1, hipStreamNonBlocking, leastPriority));
  CUDA_CHECK(hipStreamCreateWithPriority(&stream2, hipStreamNonBlocking, greaterPriority));


  CUDA_CHECK(hipEventCreate(&event));

  CUDA_CHECK(hipMemcpyAsync(d_data, h_data, size, hipMemcpyHostToDevice, stream1));
  kernel1<<<(N+255)/256, 256, 0, stream1>>>(d_data, N);

  CUDA_CHECK(hipEventRecord(event, stream1));

  CUDA_CHECK(hipStreamWaitEvent(stream2, event));

  kernel2<<<(N+255)/256, 256, 0, stream2>>>(d_data, N);

  CUDA_CHECK(hipStreamAddCallback(stream2, streamCallback, NULL, 0));

  CUDA_CHECK(hipMemcpyAsync(d_data, h_data, size, hipMemcpyDeviceToHost, stream2));

  CUDA_CHECK(hipStreamSynchronize(stream1));
  CUDA_CHECK(hipStreamSynchronize(stream2));

  CUDA_CHECK(hipHostFree(h_data));
  CUDA_CHECK(hipFree(d_data));
  CUDA_CHECK(hipStreamDestroy(stream1));
  CUDA_CHECK(hipStreamDestroy(stream2));
  CUDA_CHECK(hipEventDestroy(event));
  return 0;
}

// result:
//  nvcc -o using_callback using_callback.cu
//  ./using_callback
// Stream callback, operation completed
//  