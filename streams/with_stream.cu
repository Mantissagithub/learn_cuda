#include <iostream>
#include <hip/hip_runtime.h>
#include <time.h>

#define NUM_STREAMS 4

__global__ void vectorAdd(float* a, float* b, float* c, int n){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if(idx<n){
        c[idx] = a[idx] + b[idx];
    }
}

double get_time(){
    struct timespec tp;
    clock_gettime(CLOCK_MONOTONIC, &tp);
    return tp.tv_sec + tp.tv_nsec * 1e-9;
}

int main(){
    int size = 1024 * 1024;
    int bytes = size * sizeof(float);
    double starttime = get_time();

    hipStream_t streamsp[NUM_STREAMS];
    for(int i=0; i<NUM_STREAMS; i++){
        hipStreamCreate(&streamsp[i]);
    }

    float *h_a, *h_b, *h_c;
    hipHostMalloc((void**)&h_a, bytes, hipHostMallocDefault);
    hipHostMalloc((void**)&h_b, bytes, hipHostMallocDefault);
    hipHostMalloc((void**)&h_c, bytes, hipHostMallocDefault);

    for(int i = 0; i < size; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }

    float *d_a[NUM_STREAMS], *d_b[NUM_STREAMS], *d_c[NUM_STREAMS];
    int streamSize = size / NUM_STREAMS;
    int streamBytes = streamSize * sizeof(float);

    for(int i=0;i<NUM_STREAMS;i++){
        hipMalloc(&d_a[i], streamBytes);
        hipMalloc(&d_b[i], streamBytes);
        hipMalloc(&d_c[i], streamBytes);
    }

    for(int i=0;i<NUM_STREAMS;i++){
        int offset = i * streamSize;
        hipMemcpyAsync(d_a[i], h_a + offset, streamBytes, hipMemcpyHostToDevice, streamsp[i]);
        hipMemcpyAsync(d_b[i], h_b + offset, streamBytes, hipMemcpyHostToDevice, streamsp[i]);
        int blockSize = 256;
        int numBlocks = (streamSize + blockSize - 1) / blockSize;
        vectorAdd<<<numBlocks, blockSize, 0, streamsp[i]>>>(d_a[i], d_b[i], d_c[i], streamSize);
        hipMemcpyAsync(h_c + offset, d_c[i], streamBytes, hipMemcpyDeviceToHost, streamsp[i]);
    }

    // now comes the imp part, where we need to synchronize and wait for the streams to finish, all of them!!
    for(int i=0;i<NUM_STREAMS;i++){
        hipStreamSynchronize(streamsp[i]);
    }

    for(int i=0;i<NUM_STREAMS;i++){
        hipFree(d_a[i]);
        hipFree(d_b[i]);
        hipFree(d_c[i]);
        hipStreamDestroy(streamsp[i]);
    }

    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);

    double endtime = get_time();
    printf("Total time with streams: %.6f seconds\n", endtime - starttime);
    return 0;
}