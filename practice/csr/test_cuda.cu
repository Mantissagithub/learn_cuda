#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
    int deviceCount = 0;
    hipError_t err = hipGetDeviceCount(&deviceCount);

    if (err != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(err));
        return -1;
    }

    if (deviceCount == 0) {
        printf("no CUDA devices found.\n");
    } else {
        printf("number of CUDA devices: %d\n", deviceCount);
    }

    return 0;
}
